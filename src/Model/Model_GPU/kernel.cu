
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)
#define p 128
#define BLOCK_SIZE 128

// __global__ void compute_acc(float4 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, int n_particles)
// {
//         unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//         if (i >= n_particles ){
//                 return;
//         }
//         float3 acc_local ;
//         acc_local.x = 0.0;
//         acc_local.y = 0.0;
//         acc_local.z = 0.0;
//         for (int j = 0; j < n_particles; j++)
//         {

//                         const float diffx = positionsGPU[j].x - positionsGPU[i].x;
//                         const float diffy = positionsGPU[j].y - positionsGPU[i].y;
//                         const float diffz = positionsGPU[j].z - positionsGPU[i].z;
//                         float dij = diffx * diffx + diffy * diffy + diffz * diffz ;
//                         dij = max(1.0,dij);

//                         dij = rsqrtf(dij);
//                         dij = 10.0 * (dij * dij * dij);
                        

//                         acc_local.x += diffx * dij * positionsGPU[j].w;
//                         acc_local.y += diffy * dij * positionsGPU[j].w;
//                         acc_local.z += diffz * dij * positionsGPU[j].w;
                
//         }
//         accelerationsGPU[i].x = acc_local.x;
//         accelerationsGPU[i].y = acc_local.y;
//         accelerationsGPU[i].z = acc_local.z;
// }

__device__ float3
bodyBodyInteraction(float4 particules1, float4 particules2, float3 accel){
        float3 diff;
        diff.x = particules1.x - particules2.x;
        diff.y = particules1.y - particules2.y;
        diff.z = particules1.z - particules2.z;
        float dij = diff.x * diff.x  + diff.y  * diff.y  + diff.z  * diff.z  ;
        dij = max(1.0,dij);

        dij = rsqrtf(dij);
        dij = 10.0 * (dij * dij * dij);
        float s = dij * particules2.w;
        

        accel.x += diff.x  * s;
        accel.y += diff.y  * s;
        accel.z += diff.z  * s;
        return accel;

}
__device__ float3
tile_calculation(float4 myPosition, float3 accel){
        int i;
        extern __shared__ float4 shPosition[];
        for (i = 0; i < blockDim.x; i++){
                accel = bodyBodyInteraction(myPosition,shPosition[i],accel);
        }
        return accel;
}


__global__ void
calculate_forces(float4 * globalPosition, float4 * globalAcceleration, int n_particles){
        extern __shared__ float4 shPosition[];
        float4 myPosition;
        int i, tile;
        float3 acc = {0.0f, 0.0f, 0.0f};
        int gtid = blockIdx.x * blockDim.x + threadIdx.x;
        myPosition = globalPosition[gtid];
        for (i = 0, tile = 0 ; i < n_particles; i += p,tile++){
                int idx = tile * blockDim.x + threadIdx.x;
                shPosition[threadIdx.x] = globalPosition[idx];
                __syncthreads();
                acc = tile_calculation(myPosition, acc);
                __syncthreads();
        }
        
        float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};
        globalAcceleration[gtid] = acc4;
}

__global__ void maj_pos(float4 * positionsGPU, float4 * velocitiesGPU, float4 * accelerationsGPU,int n_particles)
{
        unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= n_particles ){
                return;
        }

        velocitiesGPU[i].x += accelerationsGPU[i].x * 2.0f;
        velocitiesGPU[i].y += accelerationsGPU[i].y * 2.0f;
        velocitiesGPU[i].z += accelerationsGPU[i].z * 2.0f;
        positionsGPU[i].x += velocitiesGPU[i].x * 0.1f;
        positionsGPU[i].y += velocitiesGPU[i].y * 0.1f;
        positionsGPU[i].z += velocitiesGPU[i].z * 0.1f;
        accelerationsGPU[i].x = 0.0f;
        accelerationsGPU[i].y = 0.0f;
        accelerationsGPU[i].z = 0.0f;

}

void update_position_cu(float4* positionsGPU, float4* velocitiesGPU, float4* accelerationsGPU, int n_particles)
{
        int nthreads = BLOCK_SIZE;
        int nblocks =  (n_particles + (nthreads -1)) / nthreads;

        calculate_forces<<<nblocks, nthreads, p>>>(positionsGPU, accelerationsGPU,n_particles);
        maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU,n_particles);
}

#endif //GALAX_MODEL_GPU

